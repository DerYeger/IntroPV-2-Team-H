#include "hip/hip_runtime.h"
#include "buddyKGVSum.cuh"

#define c(i, j) c[i * n + j]

#define T 32

__device__ int f(const int a, const int b, const int minKgv)
{
    int newA = a;
    int newB = b;

    if (a * b < minKgv) return 0;

    while(newB != 0)
    {
        if (newA > newB)
        {
            newA -= newB;
        }
        else
        {
            newB -= newA;
        }
    }

    const int kgv = a * b / newA;

    return kgv >= minKgv;
}

__global__ void calculateKGVSum(int *a, int *b, int *c, const int n, const int minKgv)
{
    __shared__ int a_tile[T][T];
    __shared__ int b_tile[T][T];

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int row = by * T + ty;
    const int col = bx * T + tx;

    const int tile_count = (T + n - 1) / T;

    int c_value = 0;

    //iterate over tiles of a
    for (int k = 0; k < tile_count; k++)
    {
        //load tile of a
        if (row < n && k * T + tx < n)
        {
            a_tile[ty][tx] = a(row, (k * T + tx));
        }
        else
        {
            a_tile[ty][tx] = 0;
        }

        __syncthreads();

        //iterate over tiles of b
        for (int m = 0; m < tile_count; m++)
        {
            //load tile of b
            if (bx * T + ty < n && m * T + tx < n)
            {
                b_tile[ty][tx] = b((bx * T + ty), (m * T + tx));
            }
            else
            {
                b_tile[ty][tx] = 0;
            }
    
            __syncthreads();
    
            //use current tile combination
            for (int g = 0; g < T; g++)
            {
                const int aig = a_tile[ty][g];
                if (aig == 0) break;
                for (int h = 0; h < T; h++)
                {
                    const int bjh =  b_tile[tx][h];
                    if (bjh == 0) break;                    
                    c_value += f(aig, bjh, minKgv);
                }
            }

            __syncthreads(); 
        }
    }

    if (row < n && col < n)
    {
        c(row, col) = c_value;
    }
}

__host__ void buddyKGVSum(int *a, int *b, const int n, const int m, const int minKgv, const int verbose, int *c)
{
    int *ad;
    int *bd;
    int *cd;

    int array_byte_size = sizeof(int) * (n * n);

    hipMalloc(&ad, array_byte_size);
    hipMalloc(&bd, array_byte_size);
    hipMalloc(&cd, array_byte_size);

    if (hipSuccess != hipGetLastError())
    {
        printf("Error allocating memory on device\n");
        exit(-1);
    }

    hipMemcpy(ad, a, array_byte_size, hipMemcpyHostToDevice);
    hipMemcpy(bd, b, array_byte_size, hipMemcpyHostToDevice);

    if (hipSuccess != hipGetLastError())
    {
        printf("Error copying memory to device\n");
        exit(-1);
    }

    const int tile_count = (T + n - 1) / T;

    // printf("Block size: %d,%d\n", T, T);
    // printf("Grid size: %d,%d\n\n", tile_count, tile_count);

    dim3 bsize(T, T);
    dim3 gsize(tile_count, tile_count);

    calculateKGVSum<<<gsize,bsize>>>(ad, bd, cd, n, minKgv);

    hipDeviceSynchronize();

    if (hipSuccess != hipGetLastError())
    {
        printf("Error executing kernel\n");
        exit(-1);
    }

    hipMemcpy(c, cd, array_byte_size, hipMemcpyDeviceToHost);

    if (hipSuccess != hipGetLastError())
    {
        printf("Error copying memory from device\n");
        exit(-1);
    }

    hipFree(ad);
    hipFree(bd);
    hipFree(cd);

    if (hipSuccess != hipGetLastError())
    {
        printf("Error freeing memory on device\n");
        exit(-1);
    }

    hipDeviceReset();
}

void initMatrices(int ** aPtr, int ** bPtr, int ** cPtr, const int n, const int seed, const int max)
{
    const int size = n * n;
    const int input_size = size * sizeof(int);
    if (NULL == *aPtr)
    {
        *aPtr = (int *) malloc(input_size);
    }
    if (NULL == *bPtr)
    {
        *bPtr = (int *) malloc(input_size);
    }
    if (NULL == *cPtr)
    {
        *cPtr = (int *) malloc(input_size);
    }

    srand(seed);
    int i;
    int j;
    int * a = *aPtr;
    int * b = *bPtr;
    for (i = 0; i < n; ++i)
    {
        for (j = 0; j < n; ++j)
        {
            a(i, j) = rand() % (max - 1 ) + 1;
        }
    }

    for (i = 0; i < n; ++i)
    {
        for (j = 0; j < n; ++j)
        {
            b(i, j) = rand() % (max - 1 ) + 1;
        }
    }
}

void freeMatrices(int ** a, int ** b, int ** c)
{
    FREE(*a);
    FREE(*b);
    FREE(*c);
}

void print(const int * a, const int n)
{
    int i;
    int j;
    for (i = 0; i < n; ++i)
    {
        for (j = 0; j < n; ++j)
        {
            printf("%d ", a(i, j));
        }
        printf("\n");
    }
}

int main(int argc, char ** argv)
{
    if (argc < 5)
    {
        printf("Program must be called with at least 4 parameters.");
        return 42;
    }
    const int n = atoi(argv[1]);
    const int m = atoi(argv[2]);
    const int minKgv = atoi(argv[3]);
    const int seed = atoi(argv[4]);
    const int verbose = argc > 5 ? atoi(argv[5]) : 0;

    int * a = NULL;
    int * b = NULL;
    int * c = NULL;
    initMatrices(&a, &b, &c, n, seed, m);
    if (0 != (2 & verbose))
    {
        print(a, n);
        printf("\n");
        print(b, n);
        printf("\n");
    }
    clock_t time = clock();
    buddyKGVSum(a, b, n, m, minKgv, verbose, c);
    time = clock() - time;

    if (0 != (1 & verbose))
    {
        print(c, n);
        printf("\n");
    }
    printf("Execution time: %f\n", (float) time / CLOCKS_PER_SEC);
    freeMatrices(&a, &b, &c);
    return 0;
}